#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_math_constants.h>
#include <stdint.h>
#include <unistd.h>
#include <png++/image.hpp>
#include <torch/torch.h>
#include "Census.cuh"
#include <ATen/ATen.h>
#include <ATen/core/TensorAccessor.h>
#include <ATen/cuda/HIPContext.h>



using namespace std;

#define TB 1024

#define DISP_MAX 256

#define COLOR_DIFF(x, i, j) (abs(x[i] - x[j]))

#define CUDA_CHECK(X)                                                          \
  do {                                                                         \
    hipError_t err = X;                                                       \
    if (err != hipSuccess) {                                                  \
      std::cerr << "CUDA error in " << __FILE__ << "(" << __LINE__             \
                << "): " << hipGetErrorString(err) << std::endl;              \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0);
  
/**********checking error after kernel has been initiated *************/

void checkCudaError(void) {
	hipError_t status = hipPeekAtLastError();
	hipError_t err = hipGetLastError();
	if (status != hipSuccess || err!=hipSuccess) {
		hipGetErrorString(status);
	}
}

/***********************************************************************/
__device__ void sort(float *x, int n)
{
	for (int i = 0; i < n - 1; i++) {
		int min = i;
		for (int j = i + 1; j < n; j++) {
			if (x[j] < x[min]) {
				min = j;
			}
		}
		float tmp = x[min];
		x[min] = x[i];
		x[i] = tmp;
	}
}
/***********************************************************************/
__global__ void ad(float *x0, float *x1, float *output, int size, int size2, int size3, int direction)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < size) {
		int d = id;
		int x = d % size3;
		d /= size3;
		int y = d % size2;
		d /= size2;
		d *= direction;

		float dist;
		if (0 <= x + d && x + d < size3) {
			int cnt = 0;
			dist = 0;
			for (int yy = y - 4; yy <= y + 4; yy++) {
				for (int xx = x - 4; xx <= x + 4; xx++) {
					if (0 <= xx && xx < size3 && 0 <= xx + d && xx + d < size3 && 0 <= yy && yy < size2) {
						int ind = yy * size3 + xx;
						dist += abs(x0[ind] - x1[ind + d]);
						cnt++;
					}
				}
			}
			dist /= cnt;
		} else {
			dist = HIP_NAN;
		}
		output[id] = dist;
	}
}
/***********************************************************************/
__global__ void census(float *x0, float *x1, float *output, int size, int num_channels, int size2, int size3, int direction)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < size) {
		int d = id;
		int x = d % size3;
		d /= size3;
		int y = d % size2;
		d /= size2;
		d *= direction;

		float dist;
		if (0 <= x + d && x + d < size3) {
			dist = 0;
			for (int i = 0; i < num_channels; i++) {
				int ind_p = (i * size2 + y) * size3 + x;
				for (int yy = y - 4; yy <= y + 4; yy++) {
					for (int xx = x - 4; xx <= x + 4; xx++) {
						if (0 <= xx && xx < size3 && 0 <= xx + d && xx + d < size3 && 0 <= yy && yy < size2) {
							int ind_q = (i * size2 + yy) * size3 + xx;
							if ((x0[ind_q] < x0[ind_p]) != (x1[ind_q + d] < x1[ind_p + d])) {
								dist++;
							}
						} else {
							dist++;
						}
					}
				}
			}
			dist /= num_channels;
		} else {
			dist = HIP_NAN;
		}
		output[id] = dist;
	}
}
/***********************************************************************/
#if 0
__global__ void add_vol(float *vol, float *cnt, float *out, int size, int size1, int size2, int size3, float ratio)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int d = id;
		int x = d % size3;
		d /= size3;
		int y = d % size2;
		d /= size2;

		int lo = floor(d * ratio);
		int hi = lo + 1;
		float alpha = (d * ratio) - lo;
		assert(0 <= lo && hi < size1);

		float val = vol[(lo * size2 + y) * size3 + x] * (1 - alpha) + vol[(hi * size2 + y) * size3 + x] * alpha;
		if (!isnan(val) && cnt[id] > 0) {
			out[id] += val;
			cnt[id] += 1;
		}
	}
}

__global__ void rho(float *x, int size, float lambda)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		x[id] = 1 - exp(-x[id] / lambda);
	}
}

#endif

__global__ void spatial_argmin(float *input, float *output, int size, int size1, int size23)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int dim23 = id % size23;
		int dim0 = id / size23;

		int argmin = 0;
		float min = HIP_INF;
		for (int i = 0; i < size1; i++) {
			float val = input[(dim0 * size1 + i) * size23 + dim23];
			if (val < min) {
				min = val;
				argmin = i;
			}
		}
		output[id] = argmin + 1;
	}
}
/***********************************************************************/
__global__ void cross(float *x0, float *out, int size, int dim2, int dim3, int L1, float tau1)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int dir = id;
		int x = dir % dim3;
		dir /= dim3;
		int y = dir % dim2;
		dir /= dim2;

		int dx = 0;
		int dy = 0;
		if (dir == 0) {
			dx = -1;
		} else if (dir == 1) {
			dx = 1;
		} else if (dir == 2) {
			dy = -1;
		} else if (dir == 3) {
			dy = 1;
		} else {
			assert(0);
		}

		int xx, yy, ind1, ind2, dist;
		ind1 = y * dim3 + x;
		for (xx = x + dx, yy = y + dy;;xx += dx, yy += dy) {
			if (xx < 0 || xx >= dim3 || yy < 0 || yy >= dim2) break;

			dist = max(abs(xx - x), abs(yy - y));
			if (dist == 1) continue;

			ind2 = yy * dim3 + xx;

			/* rule 1 */
			if (COLOR_DIFF(x0, ind1, ind2) >= tau1) break;

			/* rule 2 */
			if (dist >= L1) break;
		}
		out[id] = dir <= 1 ? xx : yy;
	}
}
/***********************************************************************/
void Cross(torch::Tensor x0, torch::Tensor out, int L1, float tau1)
{
	//
	int size_x0=sizeof(float)*x0.numel();
	int size_out=sizeof(float)*out.numel();
	
	float *x00,*out00;
	//Memory Allocation 
	int num,size2,size3;
	int L11=L1;
	float tau11=tau1;
	
	CUDA_CHECK(hipMalloc(&x00,size_x0));
	CUDA_CHECK(hipMalloc(&out00,size_out));
	

	
	// Copy data from cpu to GPU 
	CUDA_CHECK(hipMemcpy(x00  ,  x0.data_ptr<float>() ,size_x0 , hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(out00, out.data_ptr<float>() ,size_out, hipMemcpyHostToDevice));
	
	num=out.numel();
	size2=out.size(2);
	size3=out.size(3);
	
	
	cross<<<(num - 1) / TB + 1, TB>>>(
		x00,
		out00,
		num,
		size2,
		size3,
		L11, tau11);
		
	hipDeviceSynchronize();
    std::cout<<"entered cross"<<std::endl;
	checkCudaError();

	//Copy Back data from device to host 
	
	
	CUDA_CHECK(hipMemcpy(out.data_ptr<float>(), out00, size_out, hipMemcpyDeviceToHost));
	
	//Free Memory 
	hipFree(x00);
	hipFree(out00);
	//return 0;
}

/***********************************************************************/
__global__ void cbca(float *x0c, float *x1c, float *vol, float *out, int size, int dim2, int dim3, int direction)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int d = id;
		int x = d % dim3;
		d /= dim3;
		int y = d % dim2;
		d /= dim2;

		if (x + d * direction < 0 || x + d * direction >= dim3) {
			out[id] = vol[id];
		} else {
			float sum = 0;
			int cnt = 0;

			int yy_s = max(x0c[(2 * dim2 + y) * dim3 + x], x1c[(2 * dim2 + y) * dim3 + x + d * direction]);
			int yy_t = min(x0c[(3 * dim2 + y) * dim3 + x], x1c[(3 * dim2 + y) * dim3 + x + d * direction]);
			for (int yy = yy_s + 1; yy < yy_t; yy++) {
				int xx_s = max(x0c[(0 * dim2 + yy) * dim3 + x], x1c[(0 * dim2 + yy) * dim3 + x + d * direction] - d * direction);
				int xx_t = min(x0c[(1 * dim2 + yy) * dim3 + x], x1c[(1 * dim2 + yy) * dim3 + x + d * direction] - d * direction);
				for (int xx = xx_s + 1; xx < xx_t; xx++) {
					float val = vol[(d * dim2 + yy) * dim3 + xx];
					assert(!isnan(val));
					sum += val;
					cnt++;
				}
			}

			assert(cnt > 0);
			out[id] = sum / cnt;
			assert(!isnan(out[id]));
		}
	}
}
/***********************************************************************/
void CrBaCoAgg(torch::Tensor x0c, torch::Tensor x1c, torch::Tensor vol_in, torch::Tensor vol_out,  int direction)
{
	
	float *x0cc,*x1cc,*vol_inn, *vol_outt;
	int dir,num,size2,size3;
	
	int size_x0cc     = sizeof(float)*x0c.numel();
	int size_x1cc     = sizeof(float)*x1c.numel();
	int size_vol_inn  = sizeof(float)*vol_in.numel();
	int size_vol_outt = sizeof(float)*vol_out.numel();
	
	
	CUDA_CHECK(hipMalloc(&x0cc,size_x0cc));
	CUDA_CHECK(hipMalloc(&x1cc,size_x1cc));
	CUDA_CHECK(hipMalloc(&vol_inn,size_vol_inn));
	CUDA_CHECK(hipMalloc(&vol_outt,size_vol_outt));
	

	// Copy data from cpu to GPU 
	CUDA_CHECK(hipMemcpy(x0cc     ,  x0c.data_ptr<float>() ,size_x0cc , hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(x1cc     ,  x1c.data_ptr<float>() ,size_x1cc, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(vol_inn  ,  vol_in.data_ptr<float>() ,size_vol_inn, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(vol_outt ,  vol_out.data_ptr<float>() ,size_vol_outt, hipMemcpyHostToDevice));
	
	dir=direction;
	num=vol_out.numel();
	size2=vol_out.size(2);
	size3=vol_out.size(3);
	
	
	
	assert(dir == -1 or dir == 1);
	
	
	// Call to kernel 
	
	cbca<<<(num - 1) / TB + 1, TB>>>(
		x0cc,
		x1cc,
		vol_inn,
		vol_outt,
		num,
		size2,
		size3,
		dir);
		
	hipDeviceSynchronize();
	checkCudaError();
	//return 0;
	
	// Copy the necessary data and free memory 
	
	
	CUDA_CHECK(hipMemcpy(vol_out.data_ptr<float>(), vol_outt, size_vol_outt, hipMemcpyDeviceToHost));
	
	//Free Memory 
	hipFree(x0cc);
	hipFree(x1cc);
	hipFree(vol_inn);
	hipFree(vol_outt);
	
}

/***********************************************************************/
__global__ void sgm(float *x0, float *x1, float *vol, float *tmp, float *out, int dim1, int dim2, int dim3, float pi1, float pi2, float tau_so, float alpha1, float sgm_q1, float sgm_q2, int sgm_direction, int direction)
{
	int x, y, dx, dy;

	dx = dy = 0;
	if (sgm_direction <= 1) {
		y = blockIdx.x * blockDim.x + threadIdx.x;
		if (y >= dim2) {
			return;
		}
		if (sgm_direction == 0) {
			x = 0;
			dx = 1;
		} else if (sgm_direction == 1) {
			x = dim3 - 1;
			dx = -1;
		}
	} else if (sgm_direction <= 3) {
		x = blockIdx.x * blockDim.x + threadIdx.x;
		if (x >= dim3) {
			return;
		}
		if (sgm_direction == 2) {
			y = 0;
			dy = 1;
		} else if (sgm_direction == 3) {
			y = dim2 - 1;
			dy = -1;
		}
	}

	assert(dim1 <= 400);
	float tmp_curr_[400];
	float tmp_prev_[400];
	float *tmp_curr = tmp_curr_;
	float *tmp_prev = tmp_prev_;

	float min_prev = HIP_INF;
	for (; 0 <= y && y < dim2 && 0 <= x && x < dim3; x += dx, y += dy) {
		float min_curr = HIP_INF;
		for (int d = 0; d < dim1; d++) {
			int ind = (d * dim2 + y) * dim3 + x;

			if (x + d * direction < 0 ||
				x + d * direction >= dim3 || 
				y - dy < 0 || 
				y - dy >= dim2 || 
				x + d * direction - dx < 0 || 
				x + d * direction - dx >= dim3 ||
				x - dx < 0 ||
				x - dx >= dim3) {

				out[ind] += vol[ind];
				tmp_curr[d] = vol[ind];
			} else {
				int ind2 = y * dim3 + x;

				float D1 = COLOR_DIFF(x0, ind2, ind2 - dy * dim3 - dx);
				float D2 = COLOR_DIFF(x1, ind2 + d * direction, ind2 + d * direction - dy * dim3 - dx);
				float P1, P2;
				if (D1 < tau_so && D2 < tau_so) { 
					P1 = pi1; 
					P2 = (pi1 * pi2); 
				} else if (D1 > tau_so && D2 > tau_so) { 
					P1 = pi1 / (sgm_q1 * sgm_q2);
					P2 = (pi1 * pi2) / (sgm_q1 * sgm_q2);
				} else {
					P1 = pi1 / sgm_q1;
					P2 = (pi1 * pi2) / sgm_q1;
				}

				assert(min_prev != HIP_INF);
				float cost = min(tmp_prev[d], min_prev + P2);
				if (d > 0) {
					cost = min(cost, tmp_prev[d - 1] + (sgm_direction == 2 ? P1 / alpha1 : P1));
				}
				if (d < dim1 - 1) {
					cost = min(cost, tmp_prev[d + 1] + (sgm_direction == 3 ? P1 / alpha1 : P1));
				}
				float val = vol[ind] + cost - min_prev;
				out[ind] += val;
				tmp_curr[d] = val;
			}
			if (tmp_curr[d] < min_curr) {
				min_curr = tmp_curr[d];
			}
		}
		min_prev = min_curr;

		float *swap = tmp_curr;
		tmp_curr = tmp_prev;
		tmp_prev = swap;
	}
}
/***********************************************************************/
#define INDEX(dim0, dim1, dim2, dim3) \
	assert((dim1) >= 0 && (dim1) < size1 && (dim2) >= 0 && (dim2) < size2 && (dim3) >= 0 && (dim3) < size3), \
	((((dim0) * size1 + (dim1)) * size2 + (dim2)) * size3 + dim3)
/***********************************************************************/
template <int sgm_direction> 
__global__ void sgm2(float *x0, float *x1, float *input, float *output, float *tmp, float pi1, float pi2, float tau_so, float alpha1, float sgm_q1, float sgm_q2, int direction, int size1, int size2, int size3, int step)
{
	int x, y, dx, dy;
	int d = threadIdx.x;

	if (sgm_direction == 0) {
		/* right */
		x = step;
		y = blockIdx.x;
		dx = 1;
		dy = 0;
	} else if (sgm_direction == 1) {
		/* left */
		x = size2 - 1 - step;
		y = blockIdx.x;
		dx = -1;
		dy = 0;
	} else if (sgm_direction == 2) {
		/* down */
		x = blockIdx.x;
		y = step;
		dx = 0;
		dy = 1;
	} else if (sgm_direction == 3) {
		/* up */
		x = blockIdx.x;
		y = size1 - 1 - step;
		dx = 0;
		dy = -1;
	}

	if (y - dy < 0 || y - dy >= size1 || x - dx < 0 || x - dx >= size2) {
		float val = input[INDEX(0, y, x, d)];
		output[INDEX(0, y, x, d)] += val;
		tmp[d * size2 + blockIdx.x] = val;
		return;
	}

	__shared__ float output_s[400], output_min[400];

	output_s[d] = output_min[d] = tmp[d * size2 + blockIdx.x];
	__syncthreads();

	for (int i = 256; i > 0; i /= 2) {
		if (d < i && d + i < size3 && output_min[d + i] < output_min[d]) {
			output_min[d] = output_min[d + i];
		}
		__syncthreads();
	}

	int ind2 = y * size2 + x;
	float D1 = COLOR_DIFF(x0, ind2, ind2 - dy * size2 - dx);
	float D2;
	int xx = x + d * direction;
	if (xx < 0 || xx >= size2 || xx - dx < 0 || xx - dx >= size2) {
		D2 = 10;
	} else {
		D2 = COLOR_DIFF(x1, ind2 + d * direction, ind2 + d * direction - dy * size2 - dx);
	}
	float P1, P2;
	if (D1 < tau_so && D2 < tau_so) {
		P1 = pi1;
		P2 = pi2;
	} else if (D1 > tau_so && D2 > tau_so) {
		P1 = pi1 / (sgm_q1 * sgm_q2);
		P2 = pi2 / (sgm_q1 * sgm_q2);
	} else {
		P1 = pi1 / sgm_q1;
		P2 = pi2 / sgm_q1;
	}

	float cost = min(output_s[d], output_min[0] + P2);
	if (d - 1 >= 0) {
		cost = min(cost, output_s[d - 1] + (sgm_direction == 2 ? P1 / alpha1 : P1));
	}
	if (d + 1 < size3) {
		cost = min(cost, output_s[d + 1] + (sgm_direction == 3 ? P1 / alpha1 : P1));
	}

	float val = input[INDEX(0, y, x, d)] + cost - output_min[0];
	output[INDEX(0, y, x, d)] += val;
	tmp[d * size2 + blockIdx.x] = val;
}
/***********************************************************************/
void sgm2(torch::Tensor x0, torch::Tensor x1, torch::Tensor input , torch::Tensor output, torch::Tensor tmp,
     float pi1,float pi2, float tau_so, float alpha1, float sgm_q1, float sgm_q2, int direction
        )
{

	float *x00,*x11,*inputt, *outputt,*tmpp;
	float pi11,pi22,tau_soo,alpha11,sgm_q11,sgm_q22;
	int dir,size1,size2,disp_max;
	
	int size1In, size2In, size3In;
	
	int size_x00     = sizeof(float)*x0.numel();
	int size_x11     = sizeof(float)*x1.numel();
	int size_inputt  = sizeof(float)*input.numel();
	int size_outputt = sizeof(float)*output.numel();
	int size_tmpp    = sizeof(float)*tmp.numel();
	
	
	CUDA_CHECK(hipMalloc(&x00,size_x00));
	CUDA_CHECK(hipMalloc(&x11,size_x11));
	CUDA_CHECK(hipMalloc(&inputt,size_inputt));
	CUDA_CHECK(hipMalloc(&outputt,size_outputt));
	CUDA_CHECK(hipMalloc(&tmpp,size_tmpp));

	
	// Copy data from cpu to GPU 
	CUDA_CHECK(hipMemcpy(x00     ,  x0.data_ptr<float>() ,size_x00 , hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(x11     ,  x1.data_ptr<float>() ,size_x11, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(inputt  ,  input.data_ptr<float>() ,size_inputt, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(outputt ,  output.data_ptr<float>() ,size_outputt, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(tmpp    ,     tmp.data_ptr<float>() ,size_tmpp, hipMemcpyHostToDevice));
	
	// Copy variables 
	pi11     =pi1     ;
	pi22     =pi2     ;
	tau_soo  =tau_so  ;
	alpha11  =alpha1  ;
	sgm_q11  =sgm_q1  ;
	sgm_q22  =sgm_q2  ;
	dir      =direction;
	std::cout<<"pi1     : "<<pi11<<std::endl;
	std::cout<<"pi2     : "<<pi22<<std::endl;
	std::cout<<"tau_soo : "<<tau_soo<<std::endl;
	std::cout<<"alpha11 : "<<alpha11<<std::endl;
	std::cout<<"sgm_q11 : "<<sgm_q11<<std::endl;
	std::cout<<"sgm_q22 : "<<sgm_q22<<std::endl;
	std::cout<<"dir     : "<<dir<<std::endl;
	
	size1 = output.size(1)* output.size(3);
	size2 = output.size(2) * output.size(3);
	disp_max = output.size(3);
	
	std::cout<<"disparity max "<<disp_max<<std::endl;
	// input 
	size1In=input.size(1);
	size2In=input.size(2);
	size3In=input.size(3);

	for (int step = 0; step < size2In; step++) {
		sgm2<0><<<(size1 - 1) / disp_max + 1, disp_max>>>(
			x00,
			x11,
			inputt,
			outputt,
			tmpp,
			pi11, pi22, tau_soo, alpha11, sgm_q11, sgm_q22, dir,
			size1In,
			size2In,
			size3In,
			step);
	}
	
	//checkCudaError();
	for (int step = 0; step < size2In; step++) {
		sgm2<1><<<(size1 - 1) / disp_max + 1, disp_max>>>(
			x00,
			x11,
			inputt,
			outputt,
			tmpp,
			pi11, pi22, tau_soo, alpha11, sgm_q11, sgm_q22, dir,
			size1In,
			size2In,
			size3In,
			step);
	}

	//checkCudaError();
	for (int step = 0; step < size1In; step++) {
		sgm2<2><<<(size2 - 1) / disp_max + 1, disp_max>>>(
			x00,
			x11,
			inputt,
			outputt,
			tmpp,
			pi11, pi22, tau_soo, alpha11, sgm_q11, sgm_q22, dir,
			size1In,
			size2In,
			size3In,
			step);
	}

	//checkCudaError();
	for (int step = 0; step < size1In; step++) {
		sgm2<3><<<(size2 - 1) / disp_max + 1, disp_max>>>(
			x00,
			x11,
			inputt,
			outputt,
			tmpp,
			pi11, pi22, tau_soo, alpha11, sgm_q11, sgm_q22, dir,
			size1In,
			size2In,
			size3In,
			step);
	}

	checkCudaError();
	
	// copy back to host 
	CUDA_CHECK(hipMemcpy(output.data_ptr<float>(), outputt, size_outputt, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(tmp.data_ptr<float>(), tmpp, size_tmpp, hipMemcpyDeviceToHost));
	
	//Free Memory 
	hipFree(x00);
	hipFree(x11);
	hipFree(inputt);
	hipFree(outputt);
	hipFree(tmpp);
	//return 0;
}

/***********************************************************************/

template <int sgm_direction> __global__ void sgm3(float *x0, float *x1, float *input, float *output, float pi1, float pi2, float tau_so, float alpha1, float sgm_q1, float sgm_q2, int direction, int size1, int size2, int size3, int step)
{
	int x, y, dx, dy;
	int d = threadIdx.x;

	if (sgm_direction == 0) {
		/* right */
		x = step;
		y = blockIdx.x;
		dx = 1;
		dy = 0;
	} else if (sgm_direction == 1) {
		/* left */
		x = size2 - 1 - step;
		y = blockIdx.x;
		dx = -1;
		dy = 0;
	} else if (sgm_direction == 2) {
		/* down */
		x = blockIdx.x;
		y = step;
		dx = 0;
		dy = 1;
	} else if (sgm_direction == 3) {
		/* up */
		x = blockIdx.x;
		y = size1 - 1 - step;
		dx = 0;
		dy = -1;
	}

	if (y - dy < 0 || y - dy >= size1 || x - dx < 0 || x - dx >= size2) {
		output[INDEX(sgm_direction, y, x, d)] = input[INDEX(0, y, x, d)];
		return;
	}

	__shared__ float output_s[400], output_min[400];

	output_s[d] = output_min[d] = output[INDEX(sgm_direction, y - dy, x - dx, d)];
	__syncthreads();

	for (int i = 256; i > 0; i /= 2) {
		if (d < i && d + i < size3 && output_min[d + i] < output_min[d]) {
			output_min[d] = output_min[d + i];
		}
		__syncthreads();
	}

	int ind2 = y * size2 + x;
	float D1 = COLOR_DIFF(x0, ind2, ind2 - dy * size2 - dx);
	float D2;
	int xx = x + d * direction;
	if (xx < 0 || xx >= size2 || xx - dx < 0 || xx - dx >= size2) {
		D2 = 10;
	} else {
		D2 = COLOR_DIFF(x1, ind2 + d * direction, ind2 + d * direction - dy * size2 - dx);
	}
	float P1, P2;
	if (D1 < tau_so && D2 < tau_so) {
		P1 = pi1;
		P2 = pi2;
	} else if (D1 > tau_so && D2 > tau_so) {
		P1 = pi1 / (sgm_q1 * sgm_q2);
		P2 = pi2 / (sgm_q1 * sgm_q2);
	} else {
		P1 = pi1 / sgm_q1;
		P2 = pi2 / sgm_q1;
	}

	float cost = min(output_s[d], output_min[0] + P2);
	if (d - 1 >= 0) {
		cost = min(cost, output_s[d - 1] + (sgm_direction == 2 ? P1 / alpha1 : P1));
	}
	if (d + 1 < size3) {
		cost = min(cost, output_s[d + 1] + (sgm_direction == 3 ? P1 / alpha1 : P1));
	}

	output[INDEX(sgm_direction, y, x, d)] = input[INDEX(0, y, x, d)] + cost - output_min[0];
}

/***********************************************************************/
__global__ void fliplr(float *in, float *out, int size, int dim3)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int x = id % dim3;
		out[id + dim3 - 2 * x - 1] = in[id];
	}
}
/***********************************************************************/

__global__ void outlier_detection(float *d0, float *d1, float *outlier, int size, int dim3, int disp_max)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int x = id % dim3;
		int d0i = d0[id];
		if (x - d0i < 0) {
			//assert(0);
			outlier[id] = 1;
		} else if (abs(d0[id] - d1[id - d0i]) < 1.1) {
			outlier[id] = 0; /* match */
		} else {
			outlier[id] = 1; /* occlusion */
			for (int d = 0; d < disp_max; d++) {
				if (x - d >= 0 && abs(d - d1[id - d]) < 1.1) {
					outlier[id] = 2; /* mismatch */
					break;
				}
			}
		}
	}
}

/***********************************************************************/
void outlier_detection (torch::Tensor d0, torch::Tensor d1, torch::Tensor outlier, int disp_max)
{
	float *d00,*d11,*outlierr;
	int disparity,d0num,d0size2;
	int size_d00=sizeof(float)*d0.numel();
	int size_d11=sizeof(float)*d1.numel();
	int size_outlierr=sizeof(float)*outlier.numel();
	
	CUDA_CHECK(hipMalloc(&d00,size_d00));	
	CUDA_CHECK(hipMalloc(&d11,size_d11));	
	CUDA_CHECK(hipMalloc(&outlierr,size_outlierr));
		

	// Copy data from cpu to GPU 
	CUDA_CHECK(hipMemcpy(d00 , d0.data_ptr<float>() ,size_d00 , hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d11 , d1.data_ptr<float>() ,size_d11 , hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(outlierr  ,  outlier.data_ptr<float>() ,size_outlierr , hipMemcpyHostToDevice));
	disparity=disp_max;

	d0num=d0.numel();
	d0size2=d0.size(3);

	outlier_detection<<<(d0num - 1) / TB + 1, TB>>>(
		d00,
		d11,
		outlierr,
		d0num,
		d0size2,
		disparity);
		

	checkCudaError();

	//CUDA_CHECK(hipMemcpy(outlier.data_ptr<float>(), outlierr, size_outlierr, hipMemcpyDeviceToHost));

	CUDA_CHECK(hipMemcpy(d0.data_ptr<float>(), d00, size_d00, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(d1.data_ptr<float>(), d11, size_d11, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(outlier.data_ptr<float>(), outlierr, size_outlierr, hipMemcpyDeviceToHost));
	
	
	//Free Memory 
	hipFree(d00);
	hipFree(d11);
	hipFree(outlierr);
	//return 0;
}

/***********************************************************************/
#if 0

__global__ void iterative_region_voting(float *d0, float *x0c, float *x1c, float *outlier, float *d0_out, float *outlier_out, int size, int dim2, int dim3, float tau_s, float tau_h, int disp_max)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int x = id % dim3;
		int y = id / dim3;
		
		d0_out[id] = d0[id];
		outlier_out[id] = outlier[id];

		if (outlier[id] == 0) return;

		assert(disp_max < DISP_MAX);
		int hist[DISP_MAX];
		for (int i = 0; i < disp_max; i++) {
			hist[i] = 0;
		}

		int yy_s = x0c[(2 * dim2 + y) * dim3 + x];
		int yy_t = x0c[(3 * dim2 + y) * dim3 + x];
		for (int yy = yy_s + 1; yy < yy_t; yy++) {
			int xx_s = x0c[(0 * dim2 + yy) * dim3 + x];
			int xx_t = x0c[(1 * dim2 + yy) * dim3 + x];
			for (int xx = xx_s + 1; xx < xx_t; xx++) {
				if (outlier[yy * dim3 + xx] == 0) {
					hist[(int)d0[yy * dim3 + xx]]++;
				}
			}
		}

		int cnt = 0;
		int max_i = 0;
		for (int i = 0; i < disp_max; i++) {
			cnt += hist[i];
			if (hist[i] > hist[max_i]) {
				max_i = i;
			}
		}

		if (cnt > tau_s && (float)hist[max_i] / cnt > tau_h) {
			outlier_out[id] = 0;
			d0_out[id] = max_i;
		}
	}
}


#endif

/***********************************************************************/
__global__ void interpolate_mismatch(float *d0, float *outlier, float *out, int size, int dim2, int dim3)
{
	const float dir[] = {
		0	,  1,
		-0.5,  1,
		-1	,  1,
		-1	,  0.5,
		-1	,  0,
		-1	, -0.5,
		-1	, -1,
		-0.5, -1,
		0	, -1,
		0.5 , -1,
		1	, -1,
		1	, -0.5,
		1	,  0,
		1	,  0.5,
		1	,  1,
		0.5 ,  1
	};

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		if (outlier[id] != 2) {
			out[id] = d0[id];
			return;
		}

		float vals[16];
		int vals_size = 0;

		int x = id % dim3;
		int y = id / dim3;
		for (int d = 0; d < 16; d++) {
			float dx = dir[2 * d];
			float dy = dir[2 * d + 1];
			float xx = x;
			float yy = y;
			int xx_i = round(xx);
			int yy_i = round(yy);
			while (0 <= yy_i && yy_i < dim2 && 0 <= xx_i && xx_i < dim3 && outlier[yy_i * dim3 + xx_i] == 2) {
				xx += dx;
				yy += dy;
				xx_i = round(xx);
				yy_i = round(yy);
			}

			int ind = yy_i * dim3 + xx_i;
			if (0 <= yy_i && yy_i < dim2 && 0 <= xx_i && xx_i < dim3) {
				assert(outlier[ind] != 2);
				vals[vals_size++] = d0[ind];
			}
		}
		assert(vals_size > 0);
		sort(vals, vals_size);
		out[id] = vals[vals_size / 2];
	}
}

void interpolate_mismatch(torch::Tensor d0, torch::Tensor outlier, torch::Tensor out)
{
	float *d00,*outlierr,*outt;
	int size_d00=sizeof(float)*d0.numel();
	int size_outlierr=sizeof(float)*outlier.numel();
	int size_outt=sizeof(float)*out.numel();
	
	
	CUDA_CHECK(hipMalloc(&d00,size_d00));
	CUDA_CHECK(hipMalloc(&outlierr,size_outlierr));
	CUDA_CHECK(hipMalloc(&outt,size_outt));
		

	// Copy data from cpu to GPU 
	CUDA_CHECK(hipMemcpy(d00 , d0.data_ptr<float>() ,size_d00 , hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(outlierr , outlier.data_ptr<float>() ,size_outlierr , hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(outt , out.data_ptr<float>() ,size_outt , hipMemcpyHostToDevice));
	
	
	interpolate_mismatch<<<(out.numel() - 1) / TB + 1, TB>>>(
		d00,
		outlierr,
		outt,
		out.numel() ,
		out.size(2),
		out.size(3));

	checkCudaError();
	
	CUDA_CHECK(hipMemcpy(outlier.data_ptr<float>(), outlierr , size_outlierr , hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(out.data_ptr<float>(), outt , size_outt , hipMemcpyDeviceToHost));
	
	hipFree(outt);
	hipFree(outlierr);
	hipFree(d00);
	//return 1;
}

__global__ void interpolate_occlusion(float *d0, float *outlier, float *out, int size, int dim3)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		if (outlier[id] != 1) {
			out[id] = d0[id];
			return;
		}
		int x = id % dim3;

		int dx = 0;
		while (x + dx >= 0 && outlier[id + dx] != 0) {
			dx--;
		}
		if (x + dx < 0) {
			dx = 0;
			while (x + dx < dim3 && outlier[id + dx] != 0) {
				dx++;
			}
		}
		if (x + dx < dim3) {
			out[id] = d0[id + dx];
		} else {
			out[id] = d0[id];
		}
	}
}

void interpolate_occlusion(torch::Tensor d0, torch::Tensor outlier,torch::Tensor out)
{
	float *d00,*outlierr,*outt;
	int size_d00=sizeof(float)*d0.numel();
	int size_outlierr=sizeof(float)*outlier.numel();
	int size_outt=sizeof(float)*out.numel();
	
	
	CUDA_CHECK(hipMalloc(&d00,size_d00));
	CUDA_CHECK(hipMalloc(&outlierr,size_outlierr));
	CUDA_CHECK(hipMalloc(&outt,size_outt));
		

	// Copy data from cpu to GPU 
	CUDA_CHECK(hipMemcpy(d00 , d0.data_ptr<float>() ,size_d00 , hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(outlierr , outlier.data_ptr<float>() ,size_outlierr , hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(outt , out.data_ptr<float>() ,size_outt , hipMemcpyHostToDevice));
	
	interpolate_occlusion<<<(out.numel() - 1) / TB + 1, TB>>>(
		d00,
		outlierr,
		outt,
		out.numel(),
		out.size(3)
	);
	checkCudaError();
	//return 1;
	CUDA_CHECK(hipMemcpy(outlier.data_ptr<float>(), outlierr , size_outlierr , hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(out.data_ptr<float>(), outt , size_outt , hipMemcpyDeviceToHost));
	
	hipFree(outt);
	hipFree(outlierr);
	hipFree(d00);
}


#if 0

__global__ void sobel(float *x, float *g1, float *g2, int size, int dim2, int dim3)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int xx = id % dim3;
		int yy = id / dim3;

		if (1 <= yy && yy < dim2 - 1 && 1 <= xx && xx < dim3 - 1) {
			g1[id] = -x[id-dim3-1] +x[id-dim3+1] -2*x[id-1] +2*x[id+1] -x[id+dim3-1] +x[id+dim3+1];
			g2[id] = x[id-dim3-1] +2*x[id-dim3] +x[id-dim3+1] -x[id+dim3-1] -2*x[id+dim3] -x[id+dim3+1];
		} else {
			g1[id] = 0;
			g2[id] = 0;
		}
	}
}


__global__ void depth_discontinuity_adjustment(float *d0, float *dg1, float *dg2, float *xg1, float *xg2, float *out, int size, int dim3, float tau_e)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		if (abs(dg1[id]) > tau_e) {
			out[id] = xg1[id - 1] > xg1[id + 1] ? d0[id - 1] : d0[id + 1];
		} else if (abs(dg2[id]) > tau_e) {
			out[id] = xg2[id - dim3] > xg2[id + dim3] ? d0[id - dim3] : d0[id + dim3];
		} else {
			out[id] = d0[id];
		}
	}
}

#endif

__global__ void subpixel_enchancement(float *d0, float *c2, float *out, int size, int dim23, int disp_max) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int d = d0[id];
		out[id] = d;
		if (1 <= d && d < disp_max - 1) {
			float cn = c2[(d - 1) * dim23 + id];
			float cz = c2[d * dim23 + id];
			float cp = c2[(d + 1) * dim23 + id];
			float denom = 2 * (cp + cn - 2 * cz);
			if (denom > 1e-5) {
				out[id] = d - min(1.0, max(-1.0, (cp - cn) / denom));
			}
		}
	}
}

void subpixel_enchancement(torch::Tensor d0, torch::Tensor c2, torch::Tensor out, int disp_max) {

	float *d00,*c22,*outt;
	
	int size_d00=sizeof(float)*d0.numel();
	int size_c22=sizeof(float)*c2.numel();
	int size_outt=sizeof(float)*out.numel();
	
	
	CUDA_CHECK(hipMalloc(&d00,size_d00));
	CUDA_CHECK(hipMalloc(&c22,size_c22));
	CUDA_CHECK(hipMalloc(&outt,size_outt));
		

	// Copy data from cpu to GPU 
	CUDA_CHECK(hipMemcpy(d00 , d0.data_ptr<float>() ,size_d00 , hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(c22 , c2.data_ptr<float>() ,size_c22 , hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(outt , out.data_ptr<float>() ,size_outt , hipMemcpyHostToDevice));
	
	
	subpixel_enchancement<<<(out.numel() - 1) / TB + 1, TB>>>(
		d00,
		c22,
		outt,
		out.numel(),
		out.size(2)* out.size(3),
		disp_max);
	checkCudaError();
	
	CUDA_CHECK(hipMemcpy(out.data_ptr<float>(), outt , size_outt , hipMemcpyDeviceToHost));
	
	hipFree(outt);
	hipFree(c22);
	hipFree(d00);
	
}

__global__ void mean2d(float *img, float *kernel, float *out, int size, int kernel_radius, int dim2, int dim3, float alpha2)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int x = id % dim3;
		int y = id / dim3;

		float sum = 0;
		float cnt = 0;
		int i = 0;
		for (int xx = x - kernel_radius; xx <= x + kernel_radius; xx++) {
			for (int yy = y - kernel_radius; yy <= y + kernel_radius; yy++, i++) {
				if (0 <= xx && xx < dim3 && 0 <= yy && yy < dim2 && abs(img[yy * dim3 + xx] - img[y * dim3 + x]) < alpha2) {
					sum += img[yy * dim3 + xx] * kernel[i];
					cnt += kernel[i];
				}
			}
		}
		out[id] = sum / cnt;
	}
}


void mean2d(torch::Tensor img, torch::Tensor kernel, torch::Tensor out, float alpha2) {
	
	assert(kernel.size(0) % 2 == 1);
	float *imgg,*outt,*kernell;
	int size_imgg=sizeof(float)*img.numel();
	int size_outt=sizeof(float)*out.numel();
	int size_kern=sizeof(float)*kernel.numel();
	
	CUDA_CHECK(hipMalloc(&imgg,size_imgg));
	CUDA_CHECK(hipMalloc(&outt,size_outt));
	CUDA_CHECK(hipMalloc(&kernell,size_kern));
		

	// Copy data from cpu to GPU 
	CUDA_CHECK(hipMemcpy(imgg , img.data_ptr<float>() ,size_imgg , hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(outt , out.data_ptr<float>() ,size_outt , hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(kernell , kernel.data_ptr<float>() ,size_kern , hipMemcpyHostToDevice));
	
	
	mean2d<<<(out.numel() - 1) / TB + 1, TB>>>(
		imgg,
		kernell,
		outt,
		out.numel(),
		kernel.size(0) / 2,
		out.size(2),
		out.size(3),
		alpha2);
	checkCudaError();
	
	CUDA_CHECK(hipMemcpy(out.data_ptr<float>(), outt , size_outt , hipMemcpyDeviceToHost));

	//return 1;
	hipFree(imgg);
	hipFree(kernell);
	hipFree(outt);
}

__global__ void Normalize_get_norm_(float *input, float *norm, int size1, int size23, int size023)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size023) {
		int dim23 = id % size23;
		int dim0 = id / size23;

		float sum = 0.0;
		for (int dim1 = 0; dim1 < size1; dim1++) {
			float x = input[(dim0 * size1 + dim1) * size23 + dim23];
			sum += x * x;
		}
		norm[dim0 * size23 + dim23] = sum + 1e-5;
	}
}

__global__ void Normalize_forward_(float *input, float *norm, float *output, int size23, int size123, int size0123)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size0123) { 
		int dim23 = id % size23;
		int dim0 = (id / size123);
		output[id] = input[id] / sqrtf(norm[dim0 * size23 + dim23]);
	}
}


__global__ void Normalize_backward_input_(float *grad_output, float *input, float *norm, float *grad_input, int size1, int size23, int size0123)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size0123) {
		int dim0 = id;
		int dim23 = dim0 % size23;
		dim0 /= size23;
		int dim1 = dim0 % size1;
		dim0 /= size1;

		float denom = powf(norm[dim0 * size23 + dim23], 1.5);
		float deriv = (norm[dim0 * size23 + dim23] - input[id] * input[id]) / denom * grad_output[id];

		float sum = 0;
		for (int dim1_ = 0; dim1_ < size1; dim1_++) {
			if (dim1_ != dim1) {
				int ind = (dim0 * size1 + dim1_) * size23 + dim23;
				sum += input[ind] * grad_output[ind];
			}
		}
		grad_input[id] = deriv - sum * input[id] / denom;
	}
}


struct Margin2_functor {
	float margin;
	__host__ Margin2_functor(float margin_) : margin(margin_) {};
	__device__ float forward(float pos, float neg) {
		return fmaxf(0, neg - pos + margin);
	}
	__device__ float backward(float pos, float neg, int which) {
		float f = neg - pos + margin;
		if (which == 0) {
			return -1. * (f > 0);
		} else {
			return f > 0;
		}
	}
};

struct Margin2_squared_functor {
	float margin;
	__host__ Margin2_squared_functor(float margin_) : margin(margin_) {};
	__device__ float forward(float pos, float neg) {
		float d = fmaxf(0, neg - pos + margin);
		return d * d * 0.5;
	}
	__device__ float backward(float pos, float neg, int which) {
		float f = neg - pos + margin;
		if (which == 0) {
			return -f * (f > 0);
		} else {
			return f * (f > 0);
		}
	}
};

template <class Op> __global__ void Margin2_(float *input, float *tmp, float *gradInput, float margin, Op op, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		float pos = input[id * 2];
		float neg = input[id * 2 + 1];
		tmp[id] = op.forward(pos, neg);
		gradInput[id * 2] = op.backward(pos, neg, 0);
		gradInput[id * 2 + 1] = op.backward(pos, neg, 1);
	}
}


__global__ void StereoJoin_(float *input_L, float *input_R, float *output_L, float *output_R, int size1_input, int size1, int size3, int size23)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("%d \n",id);
	if (id < size23) {
		int dim3 = id % size3;
		assert(size1_input <= 128);
		float L_cache[128];
		for (int i = 0; i < size1_input; i++) {
			L_cache[i] = input_L[i * size23 + id];
		}

		for (int d = 0; d < size1; d++) {
			if (dim3 - d >= 0) {
				float sum = 0;
				for (int i = 0; i < size1_input; i++) {
					sum -= L_cache[i] * input_R[i * size23 + id - d];
				}
				output_L[d * size23 + id] = sum;
				output_R[d * size23 + id - d] = sum;
			}
		}
	}
}

/************************************************************************/
 int StereoJoin(torch::Tensor input_L, torch::Tensor input_R, torch::Tensor output_L,torch::Tensor output_R)
{
	int size23 = output_L.size(2)*output_L.size(3);
	std::cout<<"SIZE 23 "<<size23<<std::endl;
	int size1_input=input_L.size(1);
	std::cout<<"SIZE 1 IN"<<size1_input<<std::endl;
	int size1  =output_L.size(1);
	std::cout<<"SIZE 1"<<size1<<std::endl;
	int size3  =output_L.size(3);
	//std::cout<<"SIZE 3"<<size3<<std::endl;
	//sizes of data 
	int size_InputL=sizeof(float)*input_L.numel();
	int size_InputR=sizeof(float)*input_R.numel();
	int size_outputL=sizeof(float)*output_L.numel();
	int size_outputR=sizeof(float)*output_R.numel();
	std::cout<<size_InputL<<"  "<<size_InputR<<"  "<<size_outputL<<"  "<<size_outputR<<std::endl;
	float *inpL,*inpR,*outL,*outR;
	//Memory Allocation 
	
	
	CUDA_CHECK(hipMalloc(&inpL,size_InputL));
	CUDA_CHECK(hipMalloc(&inpR,size_InputR));
	CUDA_CHECK(hipMalloc(&outL,size_outputL));
	CUDA_CHECK(hipMalloc(&outR,size_outputR));
	
	std::cout<<" is contiguious "<<input_L.is_contiguous()<<std::endl;
	std::cout<<" is contiguious "<<input_R.is_contiguous()<<std::endl;
	std::cout<<" is contiguious "<<output_L.is_contiguous()<<std::endl;
	std::cout<<" is contiguious "<<output_R.is_contiguous()<<std::endl;
	
	// Copy data from cpu to GPU 
	CUDA_CHECK(hipMemcpy(inpL,input_L.data_ptr() , size_InputL, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(inpR, input_R.data_ptr(), size_InputR, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(outL, output_L.data_ptr(), size_outputL, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(outR, output_R.data_ptr(), size_outputR, hipMemcpyHostToDevice));
	
	
	StereoJoin_<<<(size23 - 1) / TB + 1, TB>>>(
		inpL,
		inpR,
		outL,
		outR,
		size1_input,
		size1,
		size3,
		size23);
		
    hipDeviceSynchronize();

    //std::cout<<"is synched "<<hipDeviceSynchronize()<<std::endl;
	checkCudaError();

	std::cout<<"entered stereo join "<<std::endl;
	
	//Copy Back data from device to host 
	
	
	CUDA_CHECK(hipMemcpy(output_L.data_ptr(), outL, size_outputL, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(output_R.data_ptr(), outR, size_outputR, hipMemcpyDeviceToHost));
	
	//Free Memory 
	hipFree(inpL);
	hipFree(inpR);
	hipFree(outL);
	hipFree(outR);
	return 0;
}
/************************************************************************/

__global__ void StereoL2R_(float *vol_L, float *vol_R, int size2, int size3, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int dim3 = id % size3;
		int dim1 = id / (size2 * size3);

		if (dim3 + dim1 >= size3) {
			vol_R[id] = HIP_INF;
		} else {
			vol_R[id] = vol_L[id + dim1];
		}
	}
}


__global__ void bilateral_filter(float *img, float *out, int size, int dim2, int dim3, int kernel_radius, float sigma1, float sigma2)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int x = id % dim3;
		int y = id / dim3;

		float sum = 0;
		float cnt = 0;
		for (int i = -kernel_radius; i <= kernel_radius; i++) {
			for (int j = -kernel_radius; j <= kernel_radius; j++) {
				int yy = y + i;
				int xx = x + j;
				if (0 <= xx && xx < dim3 && 0 <= yy && yy < dim2) {
					float color_diff = img[yy * dim3 + xx] - img[y * dim3 + x];
					float v1 = exp(-(i * i + j * j) / (2 * sigma1 * sigma1));
					float v2 = exp(-(color_diff * color_diff) / (2 * sigma2 * sigma2));
					sum += img[yy * dim3 + xx] * v1 * v2;
					cnt += v1 * v2;
				}
			}
		}
		out[id] = sum / cnt;
	}
}



__global__ void median2d(float *img, float *out, int size, int dim2, int dim3, int kernel_radius)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int x = id % dim3;
		int y = id / dim3;

		float xs[11 * 11];
		int xs_size = 0;
		for (int xx = x - kernel_radius; xx <= x + kernel_radius; xx++) {
			for (int yy = y - kernel_radius; yy <= y + kernel_radius; yy++) {
				if (0 <= xx && xx < dim3 && 0 <= yy && yy < dim2) {
					xs[xs_size++] = img[yy * dim3 + xx];
				}
			}
		}
		sort(xs, xs_size);
		out[id] = xs[xs_size / 2];
	}
}

/***********************************************************************/
void median2d(torch::Tensor img, torch::Tensor out, int kernel_size) {
	
	assert(kernel_size % 2 == 1);
	assert(kernel_size <= 11);
	
	float *imgg,*outt;
	
	int size_imgg=sizeof(float)*img.numel();
	int size_outt=sizeof(float)*out.numel();
	
	
	CUDA_CHECK(hipMalloc(&imgg,size_imgg));
	CUDA_CHECK(hipMalloc(&outt,size_outt));
		

	// Copy data from cpu to GPU 
	CUDA_CHECK(hipMemcpy(imgg , img.data_ptr<float>() ,size_imgg , hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(outt , out.data_ptr<float>() ,size_outt , hipMemcpyHostToDevice));
	
	
	median2d<<<(out.numel() - 1) / TB + 1, TB>>>(
		imgg,
		outt,
		out.numel(),
		out.size(2),
		out.size(3),
		kernel_size / 2);
	checkCudaError();
	CUDA_CHECK(hipMemcpy(out.data_ptr<float>(),outt ,size_outt , hipMemcpyDeviceToHost));
	hipFree(imgg);
	hipFree(outt);
	//return 1;
}
/***********************************************************************/
void readPNG16(torch::Tensor imgT, const char * fname)   // See later how to make it a Float Tensor 
{
	//THFloatTensor *img_ = (THFloatTensor*)luaT_checkudata(L, 1, "torch.FloatTensor");
	//const char* fname = luaL_checkstring(L, 2);

	float *img = imgT.data_ptr<float>();
	png::image<png::gray_pixel_16> image(fname);
	int width = image.get_width();
	int height = image.get_height();
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			uint16_t val = image.get_pixel(j, i);
			img[i * width + j] = val == 0 ? 0.0 : ((float)val)/256.0;
		}
	}
}
/*******************************************************************/
/*******************************************************************/
void readPNGIARPA(torch::Tensor imgT, const char * fname)
{
	//THFloatTensor *img_ = (THFloatTensor*)luaT_checkudata(L, 1, "torch.FloatTensor");
	//const char* fname = luaL_checkstring(L, 2);

	float *img = imgT.data_ptr<float>();
	png::image<png::gray_pixel_16> image(fname);
	int width = image.get_width();
	int height = image.get_height();
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			uint16_t val = image.get_pixel(j, i);
			img[i * width + j] = val == 0 ? 0.0 : ((float)val)/64.0;
		}
	}
}
/*******************************************************************/
/*******************************************************************/
void writePNG16(torch::Tensor imgT, int height, int width, const char * fname)
{
	float *img = imgT.data_ptr<float>();		
	png::image<png::gray_pixel_16> image(width, height);
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			float val = img[i * width + j];			
			image.set_pixel(j, i, (uint16_t)(val < 1e-5 ? 0 : val * 256.0));
		}
	}
	image.write(fname);
}
/*******************************************************************/

/*******************************************************************/
void writePFM(torch::Tensor imgT, const char * fname)
{
	//THFloatTensor *img_ = (THFloatTensor*)luaT_checkudata(L, 1, "torch.FloatTensor");
	//const char* fname = luaL_checkstring(L, 2);

	int height = imgT.size(0);    // size along dimension !!!!!!!!!!!!!!!!!!!
	int width = imgT.size(1);     // size along dimension !!!!!!!!!!!!!!!!!!!

	FILE *f = fopen(fname, "w");
	fprintf(f, "Pf\n%d %d\n-0.003922\n", width, height);
	fwrite(imgT.data_ptr<float>(), 4, height * width, f);
	fclose(f);
}
/*******************************************************************/
__global__ void remove_nonvisible(float *y, int size, int size3)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int x = id % size3;
		if (y[id] >= x) {
			y[id] = 0;
		}
	}
}

void remove_nonvisible(torch::Tensor disp)
{
	float * dispp;
	int size_disp=sizeof(float)*disp.numel();
	CUDA_CHECK(hipMalloc(&dispp,size_disp));
	// Copy data from cpu to GPU 
	CUDA_CHECK(hipMemcpy(dispp , disp.data_ptr<float>() ,size_disp , hipMemcpyHostToDevice));
	
	remove_occluded<<<(disp.numel() - 1) / TB + 1, TB>>>(
		dispp, 
		disp.numel(),
		disp.size(3));
	checkCudaError();
	CUDA_CHECK(hipMemcpy(disp.data_ptr<float>(),dispp ,size_disp , hipMemcpyDeviceToHost));
	hipFree(dispp);
}




__global__ void remove_occluded(float *y, int size, int size3)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int x = id % size3;
		for (int i = 1; x + i < size3; i++) {
			if (i - y[id + i] < -y[id]) {
				y[id] = 0;
				break;
			}
		}
	}
}

void remove_occluded(torch::Tensor disp)
{
	
	float *dispp;
	int size_disp=sizeof(float)*disp.numel();
	CUDA_CHECK(hipMalloc(&dispp,size_disp));
	// Copy data from cpu to GPU 
	CUDA_CHECK(hipMemcpy(dispp , disp.data_ptr<float>() ,size_disp , hipMemcpyHostToDevice));
	
	remove_occluded<<<(disp.numel() - 1) / TB + 1, TB>>>(
		dispp, 
		disp.numel(),
		disp.size(3));
	checkCudaError();
	//copy back 
	CUDA_CHECK(hipMemcpy(disp.data_ptr<float>(),dispp ,size_disp , hipMemcpyDeviceToHost));
	hipFree(dispp);
}

__global__ void remove_white(float *x, float *y, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		if (x[id] == 255) {
			y[id] = 0;
		}
	}
}

void remove_white(torch::Tensor x, torch::Tensor disp)
{
	float *xx,*dispp;
	int size_xx=sizeof(float)*x.numel();
	int size_disp=sizeof(float)*disp.numel();
	
	CUDA_CHECK(hipMalloc(&xx,size_xx));
	CUDA_CHECK(hipMalloc(&dispp,size_disp));
	
	// Copy from host 
	CUDA_CHECK(hipMemcpy(dispp , disp.data_ptr<float>() ,size_disp , hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(xx , x.data_ptr<float>() ,size_xx , hipMemcpyHostToDevice));
	
	
	remove_white<<<(disp.numel()-1) / TB + 1, TB>>>(
		xx,
		dispp,
		disp.numel());

	checkCudaError();
	CUDA_CHECK(hipMemcpy(disp.data_ptr<float>(),dispp ,size_disp , hipMemcpyDeviceToHost));
	hipFree(xx);
	hipFree(dispp);
}


__global__ void copy_fill(float *in, float *out, int size, int in_size2, int in_size3, int out_size2, int out_size3)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int out_x = id % out_size3;
		int out_y = id / out_size3;

		int in_x = out_x - (out_size3 - in_size3) / 2;
		int in_y = out_y - (out_size2 - in_size2) / 2;

		int x = min(in_size3 - 1, max(0, in_x));
		int y = min(in_size2 - 1, max(0, in_y));

		out[id] = in[y * in_size3 + x];
	}
}



void memcpy2d(float *dst, float *src, int x, int y, int win_radius, int height, int width)
{
	assert(0 <= x - win_radius);
	assert(x + win_radius <= width);
	assert(0 <= y - win_radius);
	assert(y + win_radius <= height);
	for (int i = -win_radius; i <= win_radius; i++) {
		memcpy(dst, src + (y + i) * width + x - win_radius, (win_radius * 2 + 1) * sizeof(float));
		dst += win_radius * 2 + 1;
	}
}

double random_uniform() 
{
	return ((double)rand()/(double)RAND_MAX);
}

int random_int(int a, int b)
{
	assert(a <= b);
	return floor(random_uniform() * (b - a + 1) + a);
}

double random_exp(double lambda) 
{
	double u = random_uniform();
	return -log(u) / lambda;
}


/*******************************************************************/
void subset_dataset(torch::Tensor indexT, torch::Tensor inputT, torch::Tensor outputT )
{
	long *index = indexT.data_ptr<long>();                        //  check later !!!!!!!!
	float *input = inputT.data_ptr<float>();                       //  check later !!!!!!!!
	float *output = outputT.data_ptr<float>();                     //  check later !!!!!!!!

	const int N = 200;

	int set[N];
	for (int i = 0; i < N; i++) {
		set[i] = 0;
	}

	for (int i = 0; i < indexT.numel(); i++) {   // use of numel for the total number of elements
		assert(index[i] < N);
		set[index[i]] = 1;
	}

	int i = 0;
	for (int j = 0; j < inputT.size(0); j++) {
		int im = input[j * 4];
		if (set[im]) {
			for (int k = 0; k < 4; k++) {
				output[i * 4 + k] = input[j * 4 + k];
			}
			i++;
		}
	}
}

/*******************************************************************/
void make_dataset2(torch::Tensor dispT, torch::Tensor nnzT, int img, int t)
{
	//THFloatTensor *disp_ = (THFloatTensor*)luaT_checkudata(L, 1, "torch.FloatTensor");
	//THFloatTensor *nnz_ = (THFloatTensor*)luaT_checkudata(L, 2, "torch.FloatTensor");
	//int img = luaL_checkinteger(L, 3);
	//int t = luaL_checkinteger(L, 4);

	float *disp = dispT.data_ptr<float>();
	float *nnz  = nnzT.data_ptr<float>();

	int height = dispT.size(2);
	int width =  dispT.size(3);
	int nnz_size = nnzT.numel();

	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			if (disp[i * width + j] > 0.5) {
				assert(t * 4 + 4 <= nnz_size);
				nnz[t * 4 + 0] = img;
				nnz[t * 4 + 1] = i;
				nnz[t * 4 + 2] = j;
				nnz[t * 4 + 3] = disp[i * width + j];
				t++;
			}
		}
	}
}

/* CPU implementation */
void grey2jet(torch::Tensor grey_img,torch::Tensor col_img)
{
	//THDoubleTensor *grey_img = (THDoubleTensor*)luaT_checkudata(L, 1, "torch.DoubleTensor");
	//THDoubleTensor *col_img = (THDoubleTensor*)luaT_checkudata(L, 2, "torch.DoubleTensor");

	//assert(grey_img.sizes() == 2);
	if (3 * grey_img.numel() != col_img.numel()) {
		std::cerr << "Size mismatch\n";
	}

	int height = grey_img.size(2);
	int width =  grey_img.size(3);

	float *gray_data = grey_img.data_ptr<float>();
	float *col_data  = col_img.data_ptr<float>();

	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			float val = gray_data[i * width + j] * 4;
			float r = 0, g = 0, b = 0;

			if (-0.1 <= val && val < 0.5) {
				r = 0;
				g = 0;
				b = 0.5 + val;
			} else if (0.5 <= val && val < 1.5) {
				r = 0;
				g = val - 0.5;
				b = 1;
			} else if (1.5 <= val && val < 2.5) {
				r = val - 1.5;
				g = 1;
				b = 1 - (val - 1.5);
			} else if (2.5 <= val && val < 3.5) {
				r = 1;
				g = 1 - (val - 2.5);
				b = 0;
			} else if (3.5 <= val && val <= 4.1) {
				r = 1 - (val - 3.5);
				g = 0;
				b = 0;
			} else {
				//printf("val = %f\n", val);
				assert(0);
			}

			col_data[(0 * height + i) * width + j] = r;
			col_data[(1 * height + i) * width + j] = g;
			col_data[(2 * height + i) * width + j] = b;
		}
	}
}
